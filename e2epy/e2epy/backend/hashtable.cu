#include <iostream>
#include <inttypes.h>
#include <hip/hip_runtime.h>

#include "common.h"
#include "io.h"
#include "kernels.h"
#include "hashtable.h"

HashTable::HashTable(
    const uint64_t *keys, const bool keys_on_device, const int size, const int capacity)
{
  size_m = size;
  capacity_m = capacity;

  cuda_errchk(
      hipMalloc(&keys_m, capacity*sizeof(uint64_t)));
  cuda_errchk(
      hipMemset(keys_m, 0xFF, capacity*sizeof(uint64_t)));
  cuda_errchk(
      hipMalloc(&values_m, capacity*sizeof(uint32_t)));
  cuda_errchk(
      hipMemset(values_m, 0, capacity*sizeof(uint32_t)));

  uint64_t *keys_d;
  if (!keys_on_device)
  {
    cuda_errchk(
        hipMalloc(&keys_d, size*sizeof(uint64_t)));
    cuda_errchk(
        hipMemcpy(keys_d, keys, size*sizeof(uint64_t), hipMemcpyHostToDevice));
  }

  // Synchronize because hipMemset is asynchronous with respect to host
  cuda_errchk(hipDeviceSynchronize());

  kernels::initialize_hashtable(
      keys_m, capacity, keys_on_device ? keys : keys_d, size);

  if (!keys_on_device)
  {
    cuda_errchk(hipFree(keys_d));
  }
}

HashTable::~HashTable()
{
  cuda_errchk(hipFree(keys_m));
  cuda_errchk(hipFree(values_m));
}

void HashTable::count(const char *filename, const int header_length, const int read_length, 
    const int reads_per_chunk, const int kmer_size)
{
  FastaReader reader(filename);
  while (!reader.done())
  {
    char *reads;
    int num_reads = reader.read_chunk(&reads, reads_per_chunk, header_length, read_length);

    char *reads_d;
    cuda_errchk(
        hipMalloc(&reads_d, read_length*num_reads*sizeof(char)));
    cuda_errchk(
        hipMemcpy(reads_d, reads, read_length*num_reads*sizeof(char), hipMemcpyHostToDevice));

    kernels::count_reads(keys_m, values_m, capacity_m, reads_d, num_reads, read_length, kmer_size);
    //cuda_errchk(hipDeviceSynchronize());

    delete[] reads;
    cuda_errchk(hipFree(reads_d));
  }
}
