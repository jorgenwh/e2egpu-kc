#include <iostream>
#include <inttypes.h>
#include <hip/hip_runtime.h>

#include "common.h"
#include "io.h"
#include "kernels.h"
#include "hashtable.h"

HashTable::HashTable(
    const uint64_t *keys, const bool keys_on_device, const int size, const int capacity)
{
  size_m = size;
  capacity_m = capacity;

  cuda_errchk(
      hipMalloc(&keys_m, capacity*sizeof(uint64_t)));
  cuda_errchk(
      hipMemset(keys_m, 0xFF, capacity*sizeof(uint64_t)));
  cuda_errchk(
      hipMalloc(&values_m, capacity*sizeof(uint32_t)));
  cuda_errchk(
      hipMemset(values_m, 0, capacity*sizeof(uint32_t)));

  uint64_t *keys_d;
  if (!keys_on_device)
  {
    cuda_errchk(
        hipMalloc(&keys_d, size*sizeof(uint64_t)));
    cuda_errchk(
        hipMemcpy(keys_d, keys, size*sizeof(uint64_t), hipMemcpyHostToDevice));
  }

  // Synchronize because hipMemset is asynchronous with respect to host
  cuda_errchk(hipDeviceSynchronize());

  kernels::initialize_hashtable(
      keys_m, capacity, keys_on_device ? keys : keys_d, size);

  if (!keys_on_device)
  {
    cuda_errchk(hipFree(keys_d));
  }
}

HashTable::~HashTable()
{
  cuda_errchk(hipFree(keys_m));
  cuda_errchk(hipFree(values_m));
}

void HashTable::count(const char *filename, const int header_length, const int read_length, 
    const int reads_per_chunk, const int kmer_size)
{
  int i = 0;
  FastaReader reader(filename);
  while (!reader.done())
  {
    char *reads;
    int num_reads = reader.read_chunk(&reads, reads_per_chunk, header_length, read_length);

    char *reads_d;
    cuda_errchk(
        hipMalloc(&reads_d, read_length*num_reads*sizeof(char)));
    cuda_errchk(
        hipMemcpy(reads_d, reads, read_length*num_reads*sizeof(char), hipMemcpyHostToDevice));

    //kernels::count_reads(
        //keys_m, values_m, capacity_m, reads_d, num_reads, read_length, kmer_size);
    kernels::count_reads_single(
        keys_m, values_m, capacity_m, reads_d, num_reads, read_length, kmer_size);

    delete[] reads;
    cuda_errchk(hipFree(reads_d));

    i++;
    std::cout << "Chunk " << i << "\r";
  }
  std::cout << "Chunk " << i << "\n";
}


void HashTable::count_fasta_chunks(const char *filename, const int chunk_size, 
    const int header_length, const int read_length, const int kmer_size)
{
  int i = 0;
  FastaReader reader(filename);
  while (!reader.done())
  {
    char *chunk;
    int bytes_read = reader.read_raw_chunk(&chunk, chunk_size);

    kernels::count_raw_reads(chunk, bytes_read, header_length, read_length, kmer_size);

    delete[] chunk;

    i++;
    std::cout << "Chunk " << i << "\r";
  }
  std::cout << "Chunk " << i << "\n";
}

void HashTable::lookup(const uint64_t *keys, uint32_t *values, int size) const
{
  uint64_t *keys_d;
  uint32_t *values_d;
  cuda_errchk(hipMalloc(&keys_d, size*sizeof(uint64_t)));
  cuda_errchk(hipMalloc(&values_d, size*sizeof(uint32_t)));
  cuda_errchk(hipMemcpy(keys_d, keys, size*sizeof(uint64_t), hipMemcpyHostToDevice));

  kernels::lookup(keys_m, values_m, capacity_m, keys_d, values_d, size);

  cuda_errchk(hipMemcpy(values, values_d, size*sizeof(uint32_t), hipMemcpyDeviceToHost)); 
  cuda_errchk(hipFree(keys_d));
  cuda_errchk(hipFree(values_d));
}
