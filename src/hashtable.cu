#include <iostream>
#include <inttypes.h>
#include <hip/hip_runtime.h>

#include "common.h"
#include "io.h"
#include "kernels.h"
#include "hashtable.h"

HashTable::HashTable(
    const uint64_t *keys, const bool keys_on_device, const int size, const int capacity)
{
  size_m = size;
  capacity_m = capacity;

  cuda_errchk(
      hipMalloc(&keys_m, capacity*sizeof(uint64_t)));
  cuda_errchk(
      hipMemset(keys_m, 0xFF, capacity*sizeof(uint64_t)));
  cuda_errchk(
      hipMalloc(&values_m, capacity*sizeof(uint32_t)));
  cuda_errchk(
      hipMemset(values_m, 0, capacity*sizeof(uint32_t)));

  uint64_t *keys_d;
  if (!keys_on_device)
  {
    cuda_errchk(
        hipMalloc(&keys_d, size*sizeof(uint64_t)));
    cuda_errchk(
        hipMemcpy(keys_d, keys, size*sizeof(uint64_t), hipMemcpyHostToDevice));
  }

  // Synchronize because hipMemset is asynchronous with respect to host
  cuda_errchk(hipDeviceSynchronize());

  kernels::initialize_hashtable(
      keys_m, capacity, keys_on_device ? keys : keys_d, size);

  if (!keys_on_device)
  {
    cuda_errchk(hipFree(keys_d));
  }
}

HashTable::~HashTable()
{
  cuda_errchk(hipFree(keys_m));
  cuda_errchk(hipFree(values_m));
}

void HashTable::count(const char *filename, const int header_length, const int read_length, 
    const int reads_per_chunk, const int kmer_size)
{
  FastaReader reader(filename);
  while (!reader.done())
  {
    char *reads;
    int num_reads = reader.read_chunk(&reads, reads_per_chunk, header_length, read_length);

    char *reads_d;
    cuda_errchk(
        hipMalloc(&reads_d, read_length*num_reads*sizeof(char)));
    cuda_errchk(
        hipMemcpy(reads_d, reads, read_length*num_reads*sizeof(char), hipMemcpyHostToDevice));

    kernels::count_reads(keys_m, values_m, capacity_m, reads_d, num_reads, read_length, kmer_size);
    //cuda_errchk(hipDeviceSynchronize());

    delete[] reads;
    cuda_errchk(hipFree(reads_d));
  }
}

void HashTable::get(const uint64_t *keys, uint32_t *values, int size) const
{
  uint64_t *keys_d;
  uint32_t *values_d;
  cuda_errchk(hipMalloc(&keys_d, size*sizeof(uint64_t)));
  cuda_errchk(hipMalloc(&values_d, size*sizeof(uint32_t)));
  cuda_errchk(hipMemcpy(keys_d, keys, size*sizeof(uint64_t), hipMemcpyHostToDevice));

  kernels::lookup(keys_m, values_m, capacity_m, keys_d, values_d, size);

  cuda_errchk(hipMemcpy(values, values_d, size*sizeof(uint32_t), hipMemcpyDeviceToHost)); 
  cuda_errchk(hipFree(keys_d));
  cuda_errchk(hipFree(values_d));
}
