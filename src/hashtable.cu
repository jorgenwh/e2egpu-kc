#include <iostream>
#include <inttypes.h>
#include <hip/hip_runtime.h>

#include "common.h"
#include "kernels.h"
#include "hashtable.h"

HashTable::HashTable(
    const uint64_t *keys, const bool keys_on_device, const int size, const int capacity)
{
  size_m = size;
  capacity_m = capacity;

  cuda_errchk(
      hipMalloc(&keys_m, capacity*sizeof(uint64_t)));
  cuda_errchk(
      hipMemset(keys_m, 0xFF, capacity*sizeof(uint64_t)));
  cuda_errchk(
      hipMalloc(&values_m, capacity*sizeof(uint32_t)));
  cuda_errchk(
      hipMemset(values_m, 0, capacity*sizeof(uint32_t)));

  uint64_t *keys_d;
  if (!keys_on_device)
  {
    cuda_errchk(
        hipMalloc(&keys_d, size*sizeof(uint64_t)));
    cuda_errchk(
        hipMemcpy(keys_d, keys, size*sizeof(uint64_t), hipMemcpyHostToDevice));
  }

  // Synchronize because hipMemset is asynchronous with respect to host
  cuda_errchk(hipDeviceSynchronize());

  kernels::initialize_hashtable(
      keys_m, values_m, keys_on_device ? keys : keys_d, size, capacity);

  if (!keys_on_device)
  {
    cuda_errchk(hipFree(keys_d));
  }
}

HashTable::~HashTable()
{
  cuda_errchk(hipFree(keys_m));
  cuda_errchk(hipFree(values_m));
}
